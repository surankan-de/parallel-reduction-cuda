#include "hip/hip_runtime.h"
#include <iostream>
#include<hip/hip_runtime.h>
#include <chrono>
#include <numeric> 

// REDUCTION 3 – First Add During Load
__global__ void first_add_load(int *g_in_data, int *g_out_data){
    extern __shared__ int sdata[];  // stored in the shared memory

    // Each thread loading one element from global onto shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;
    sdata[tid] = max(g_in_data[i] , g_in_data[i+blockDim.x]);
    __syncthreads();

    // Reduction method -- occurs in shared memory
    for(unsigned int s = blockDim.x/2; s > 0; s >>= 1){
        // check out the reverse loop above
        if (tid < s){   // then, we check tid to do our computation
            sdata[tid] =max(sdata[tid], sdata[tid + s]);
        }
        __syncthreads();
    }
    if (tid == 0){
        g_out_data[blockIdx.x] = sdata[0];
    }
}

// I hope to use this main file for all of the reduction files
int main(){
    int n = 1<<22; // Increase to about 4M elements
    size_t bytes = n * sizeof(int);

    // Host/CPU arrays
    int *host_input_data = new int[n];
    int *host_output_data = new int[(n + 255) / 256]; // to have sufficient size for output array

    // Device/GPU arrays
    int *dev_input_data, *dev_output_data;

    // Init data
    srand(42); // Fixed seed
    for (int i = 0; i < n; i++){
        host_input_data[i] = rand() % 100;
    }

    // Allocating memory on GPU for device arrays
    hipMalloc(&dev_input_data, bytes);
    hipMalloc(&dev_output_data, (n + 255) / 256 * sizeof(int));

    // Copying our data onto the device (GPU)
    hipMemcpy(dev_input_data, host_input_data, bytes, hipMemcpyHostToDevice);

    int blockSize = 256; // number of threads per block

    auto start = std::chrono::high_resolution_clock::now(); // start timer

    // Launch Kernel and Synchronize threads
    int num_blocks = (n + (2 * blockSize) - 1) / (2 * blockSize);   // Modifying this to account for the fact that 1 thread accesses 2 elements
    hipError_t err;
    first_add_load<<<num_blocks, blockSize, blockSize * sizeof(int)>>>(dev_input_data, dev_output_data);
    err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
    }
    hipDeviceSynchronize();

    auto stop = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start).count() / 1000.0; // duration in milliseconds with three decimal points

    // Copying data back to the host (CPU)
    hipMemcpy(host_output_data, dev_output_data, (n + 255) / 256 * sizeof(int), hipMemcpyDeviceToHost);

    // Final reduction on the host
    int finalResult = host_output_data[0];
    for (int i = 1; i < (n + 255) / 256; ++i) {
        finalResult =max(finalResult, host_output_data[i]);
    }

    // CPU Summation for verification
    int cpuResult = *std::max_element(host_input_data, host_input_data + n);
    if (cpuResult == finalResult) {
        std::cout << "\033[32m"; // Set text color to green
        std::cout << "Verification successful: GPU result matches CPU result.\n";
        std::cout << "GPU Result: " << finalResult << ", CPU Result: " << cpuResult << std::endl;
    } else {
        std::cout << "\033[31m"; // Set text color to red
        std::cout << "Verification failed: GPU result (" << finalResult << ") does not match CPU result (" << cpuResult << ").\n";
        std::cout << "GPU Result: " << finalResult << ", CPU Result: " << cpuResult << std::endl;
    }
    std::cout << "\033[0m"; // Reset text color to default

    double bandwidth = (duration > 0) ? (bytes / duration / 1e6) : 0; // computed in GB/s, handling zero duration
    std::cout << "Reduced result: " << finalResult << std::endl;
    std::cout << "Time elapsed: " << duration << " ms" << std::endl;
    std::cout << "Effective bandwidth: " << bandwidth << " GB/s" << std::endl;

    // Freeing memory
    hipFree(dev_input_data);
    hipFree(dev_output_data);
    delete[] host_input_data;
    delete[] host_output_data;
}