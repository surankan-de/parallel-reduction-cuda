#include <iostream>
#include<hip/hip_runtime.h>
#include <chrono>
#include <numeric> 

// REDUCTION 1 – Interleaved Addressing without branch divergence
__global__ void interleaved_addressing_2(int *g_in_data, int *g_out_data){
    extern __shared__ int sdata[];  // stored in the shared memory

    
    unsigned int tid = threadIdx.x; //thread id
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; //index to take 
    sdata[tid] = g_in_data[i];
    __syncthreads();

    // Reduction method -- occurs in shared memory
    for(unsigned int s = 1; s < blockDim.x; s *= 2){
        // note the stride as s *= 2 : this causes the interleaving addressing
        int index = 2 * s * tid;    // mod removed and we don't need a diverging branch from the if condition so no more waits
        if (index + s < blockDim.x)
        {
            sdata[index] += sdata[index + s];   // s is the offset to combine
        }
        __syncthreads();
    }
    if (tid == 0){
        g_out_data[blockIdx.x] = sdata[0];
    }
}

// I hope to use this main file for all of the reduction files
int main(){
    int dataSize = 1 << 22;
size_t byteSize = dataSize * sizeof(int);

int *hostInput = new int[dataSize];
int *hostOutput = new int[(dataSize + 255) / 256];

int *deviceInput, *deviceOutput;

srand(42);
for (int i = 0; i < dataSize; i++) {
    hostInput[i] = rand() % 100;
}

hipMalloc(&deviceInput, byteSize);
hipMalloc(&deviceOutput, (dataSize + 255) / 256 * sizeof(int));

hipMemcpy(deviceInput, hostInput, byteSize, hipMemcpyHostToDevice);

int threadCount = 256;

auto startTime = std::chrono::high_resolution_clock::now();

int blockCount = (dataSize + threadCount - 1) / threadCount;
hipError_t error;
interleaved_addressing_2<<<blockCount, threadCount, threadCount * sizeof(int)>>>(deviceInput, deviceOutput);
error = hipGetLastError();
if (error != hipSuccess) {
    std::cerr << "CUDA error: " << hipGetErrorString(error) << std::endl;
}
hipDeviceSynchronize();

auto endTime = std::chrono::high_resolution_clock::now();
double elapsedTime = std::chrono::duration_cast<std::chrono::microseconds>(endTime - startTime).count() / 1000.0;

hipMemcpy(hostOutput, deviceOutput, (dataSize + 255) / 256 * sizeof(int), hipMemcpyDeviceToHost);

int sumGPU = hostOutput[0];
for (int i = 1; i < (dataSize + 255) / 256; ++i) {
    sumGPU += hostOutput[i];
}

int sumCPU = std::accumulate(hostInput, hostInput + dataSize, 0);
if (sumCPU == sumGPU) {
    std::cout << "\033[32m";
    std::cout << "Verification successful: GPU result matches CPU result.\n";
    std::cout << "GPU Result: " << sumGPU << ", CPU Result: " << sumCPU << std::endl;
} else {
    std::cout << "\033[31m";
    std::cout << "Verification failed: GPU result (" << sumGPU << ") does not match CPU result (" << sumCPU << ").\n";
    std::cout << "GPU Result: " << sumGPU << ", CPU Result: " << sumCPU << std::endl;
}
std::cout << "\033[0m";

double memoryBandwidth = (elapsedTime > 0) ? (byteSize / elapsedTime / 1e6) : 0;
std::cout << "Reduced result: " << sumGPU << std::endl;
std::cout << "Time elapsed: " << elapsedTime << " ms" << std::endl;
std::cout << "Effective bandwidth: " << memoryBandwidth << " GB/s" << std::endl;

hipFree(deviceInput);
hipFree(deviceOutput);
delete[] hostInput;
delete[] hostOutput;
}