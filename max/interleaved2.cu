#include "hip/hip_runtime.h"
#include <iostream>
#include<hip/hip_runtime.h>
#include <chrono>
#include <numeric> 

// REDUCTION 1 – Interleaved Addressing without branch divergence
__global__ void interleaved_addressing_2(int *g_in_data, int *g_out_data){
    extern __shared__ int sdata[];  // stored in the shared memory

    
    unsigned int tid = threadIdx.x; //thread id
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; //index to take 
    sdata[tid] = g_in_data[i];
    __syncthreads();

    // Reduction method -- occurs in shared memory
    for(unsigned int s = 1; s < blockDim.x; s *= 2){
        // note the stride as s *= 2 : this causes the interleaving addressing
        int index = 2 * s * tid;    // mod removed and we don't need a diverging branch from the if condition so no more waits
        if (index + s < blockDim.x)
        {
            sdata[index] =max(sdata[index], sdata[index + s]);   // s is the offset to combine
        }
        __syncthreads();
    }
    if (tid == 0){
        g_out_data[blockIdx.x] = sdata[0];
    }
}

// I hope to use this main file for all of the reduction files
int main(){
    int n = 1 << 22; // Increase to about 4M elements
    size_t bytes = n * sizeof(int);

    // Host/CPU arrays
    int *host_input_data = new int[n];
    int *host_output_data = new int[(n + 255) / 256]; // to have sufficient size for output array

    // Device/GPU arrays
    int *dev_input_data, *dev_output_data;

    // Init data
    srand(42); // Fixed seed
    for (int i = 0; i < n; i++){
        host_input_data[i] = rand() % 100;
    }

    // Allocating memory on GPU for device arrays
    hipMalloc(&dev_input_data, bytes);
    hipMalloc(&dev_output_data, (n + 255) / 256 * sizeof(int));

    // Copying our data onto the device (GPU)
    hipMemcpy(dev_input_data, host_input_data, bytes, hipMemcpyHostToDevice);

    int blockSize = 256; // number of threads per block

    auto start = std::chrono::high_resolution_clock::now(); // start timer

    // Launch Kernel and Synchronize threads
    int num_blocks = (n + blockSize - 1) / blockSize;
    hipError_t err;
    interleaved_addressing_2<<<num_blocks, blockSize, blockSize * sizeof(int)>>>(dev_input_data, dev_output_data);
    err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
    }
    hipDeviceSynchronize();

    auto stop = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start).count() / 1000.0; // duration in milliseconds with three decimal points

    // Copying data back to the host (CPU)
    hipMemcpy(host_output_data, dev_output_data, (n + 255) / 256 * sizeof(int), hipMemcpyDeviceToHost);

    // Final reduction on the host
    int finalResult = host_output_data[0];
    for (int i = 1; i < (n + 255) / 256; ++i) {
        finalResult =max(finalResult,host_output_data[i]);
    }

    // CPU Summation for verification
    int cpuResult = *std::max_element(host_input_data, host_input_data + n);
    if (cpuResult == finalResult) {
        std::cout << "\033[32m"; // Set text color to green
        std::cout << "Verification successful: GPU result matches CPU result.\n";
        std::cout << "GPU Result: " << finalResult << ", CPU Result: " << cpuResult << std::endl;
    } else {
        std::cout << "\033[31m"; // Set text color to red
        std::cout << "Verification failed: GPU result (" << finalResult << ") does not match CPU result (" << cpuResult << ").\n";
        std::cout << "GPU Result: " << finalResult << ", CPU Result: " << cpuResult << std::endl;
    }
    std::cout << "\033[0m"; // Reset text color to default

    double bandwidth = (duration > 0) ? (bytes / duration / 1e6) : 0; // computed in GB/s, handling zero duration
    std::cout << "Reduced result: " << finalResult << std::endl;
    std::cout << "Time elapsed: " << duration << " ms" << std::endl;
    std::cout << "Effective bandwidth: " << bandwidth << " GB/s" << std::endl;

    // Freeing memory
    hipFree(dev_input_data);
    hipFree(dev_output_data);
    delete[] host_input_data;
    delete[] host_output_data;
}