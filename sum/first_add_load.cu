#include <iostream>
#include<hip/hip_runtime.h>
#include <chrono>
#include <numeric> 

//load first then start
__global__ void first_add_load(int *g_in_data, int *g_out_data){
    extern __shared__ int sdata[];  // stored in the shared memory

    // Each thread loading one element from global onto shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;
    sdata[tid] = g_in_data[i] + g_in_data[i+blockDim.x];
    __syncthreads();

    // Reduction method -- occurs in shared memory
    for(unsigned int s = blockDim.x/2; s > 0; s >>= 1){
        // check out the reverse loop above
        if (tid < s){   // then, we check tid to do our computation
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }
    if (tid == 0){
        g_out_data[blockIdx.x] = sdata[0];
    }
}


int main(){
    int dataSize = 1 << 22;
size_t byteSize = dataSize * sizeof(int);

int *hostInput = new int[dataSize];
int *hostOutput = new int[(dataSize + 255) / 256];

int *deviceInput, *deviceOutput;

srand(42);
for (int i = 0; i < dataSize; i++) {
    hostInput[i] = rand() % 100;
}

hipMalloc(&deviceInput, byteSize);
hipMalloc(&deviceOutput, (dataSize + 255) / 256 * sizeof(int));

hipMemcpy(deviceInput, hostInput, byteSize, hipMemcpyHostToDevice);

int threadCount = 256;

auto startTime = std::chrono::high_resolution_clock::now();

int blockCount = (dataSize + (2 * threadCount) - 1) / (2 * threadCount);
hipError_t error;
first_add_load<<<blockCount, threadCount, threadCount * sizeof(int)>>>(deviceInput, deviceOutput);
error = hipGetLastError();
if (error != hipSuccess) {
    std::cerr << "CUDA error: " << hipGetErrorString(error) << std::endl;
}
hipDeviceSynchronize();

auto endTime = std::chrono::high_resolution_clock::now();
double elapsedTime = std::chrono::duration_cast<std::chrono::microseconds>(endTime - startTime).count() / 1000.0;

hipMemcpy(hostOutput, deviceOutput, (dataSize + 255) / 256 * sizeof(int), hipMemcpyDeviceToHost);

int sumGPU = hostOutput[0];
for (int i = 1; i < (dataSize + 255) / 256; ++i) {
    sumGPU += hostOutput[i];
}

int sumCPU = std::accumulate(hostInput, hostInput + dataSize, 0);
if (sumCPU == sumGPU) {
    std::cout << "\033[32m";
    std::cout << "Verification successful: GPU result matches CPU result.\n";
    std::cout << "GPU Result: " << sumGPU << ", CPU Result: " << sumCPU << std::endl;
} else {
    std::cout << "\033[31m";
    std::cout << "Verification failed: GPU result (" << sumGPU << ") does not match CPU result (" << sumCPU << ").\n";
    std::cout << "GPU Result: " << sumGPU << ", CPU Result: " << sumCPU << std::endl;
}
std::cout << "\033[0m";

double memoryBandwidth = (elapsedTime > 0) ? (byteSize / elapsedTime / 1e6) : 0;
std::cout << "Reduced result: " << sumGPU << std::endl;
std::cout << "Time elapsed: " << elapsedTime << " ms" << std::endl;
std::cout << "Effective bandwidth: " << memoryBandwidth << " GB/s" << std::endl;

hipFree(deviceInput);
hipFree(deviceOutput);
delete[] hostInput;
delete[] hostOutput;

}